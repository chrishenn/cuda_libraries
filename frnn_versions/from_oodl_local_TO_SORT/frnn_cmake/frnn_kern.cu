#include "hip/hip_runtime.h"
#include <torch/types.h>

#include <stdio.h>
#include <iostream>


// load a bin's pts, where we are interested in [y, x, z, scale, ptid] at columns
// [0,1,2,4,5] in pts. A bin starts at bin_start in bin_ptids and includes contiguous
// rows, of number bin_size. bin_ptids gives indexes into pts.
template <typename scalar_t>
__device__ void LoadBin(const scalar_t* pts, const int* bin_ptids, float* bin,
                        int bin_start, int bin_size,
                        const int pt_size1,
                        const int bin_stride)
{
    for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < bin_size; i += blockDim.x * blockDim.y)
    {
        int pt_id = bin_ptids[ bin_start + i ];

        bin[i * bin_stride + 0] = float( pts[pt_id * pt_size1 + 0] );
        bin[i * bin_stride + 1] = float( pts[pt_id * pt_size1 + 1] );
        bin[i * bin_stride + 2] = float( pts[pt_id * pt_size1 + 2] );
        bin[i * bin_stride + 3] = float( pts[pt_id * pt_size1 + 4] );
        bin[i * bin_stride + 4] = float( pt_id );
    }
}

// compare two bins worth of pts for frnn criteria; write to edges. The samebin flag indicates that
// bin_a and bin_b are the same; avoid redundant edges by enforcing ptid_diff > 0 in this case.
__device__ void CompareBin(
        long* edges, int* glob_counts,
        float* bin_a, float* bin_b, int bin_Awidth, int bin_Bwidth,
        float radius, float scale_radius,
        int bin_stride, bool samebin )
{
    int thid = threadIdx.y * blockDim.x + threadIdx.x;
    int block_threads = (blockDim.x * blockDim.y);

    for (int ia = thid; ia < bin_Awidth; ia += block_threads)
    {
        float ay =      bin_a[ia * bin_stride + 0];
        float ax =      bin_a[ia * bin_stride + 1];
        float az =      bin_a[ia * bin_stride + 2];
        float as =      bin_a[ia * bin_stride + 3];
        float a_ptid =  bin_a[ia * bin_stride + 4];

        for (int ib = 0; ib < bin_Bwidth; ib++)
        {
            float by =      bin_b[ib * bin_stride + 0];
            float bx =      bin_b[ib * bin_stride + 1];
            float bz =      bin_b[ib * bin_stride + 2];
            float bs =      bin_b[ib * bin_stride + 3];
            float b_ptid =  bin_b[ib * bin_stride + 4];

            float diffx = bx - ax;
            float diffy = by - ay;
            float diffz = bz - az;

            float dist = sqrtf( diffx*diffx + diffy*diffy + diffz*diffz );

            float ptid_diff = a_ptid - b_ptid;

            // a_ptid is not b_ptid
            bool check0 = fabsf(ptid_diff) > 0.2;

            // if same bin a and b, no redundant edges written
            bool check1 = (!samebin) || (ptid_diff > 0);

            // frnn criteria in linear, scale space
            bool check2 = dist < ( radius * sqrtf(as*bs) ) && fabsf(logf(as) - logf(bs)) < scale_radius;

            // write valid edge
            if (check0 && check1 && check2)
            {
                int thread_i = atomicAdd(glob_counts, 2);
                edges[thread_i + 0] = long(a_ptid);
                edges[thread_i + 1] = long(b_ptid);
            }
        }
    }

}

// each block loads two bins into shared mem for comparison, including comparing the central bin at column 0 to itself.
template <typename scalar_t>
__global__ void frnn_main_kernel(

        const   int*    neighbors,
        const   int*     bin_counts,
        const   int*     bin_offsets,
        const   int*     bin_ptids,
        const   scalar_t*   pts,

        int*      glob_counts,
        long*     edges,

        const float    radius,
        const float    scale_radius,

        const int      max_binsize,
        const int      bin_stride,

        const int      neighb_size1,
        const int      pt_size1
){

    extern __shared__ float s[];
    float* bin_a = s;
    float* bin_b = (float*)& bin_a[ max_binsize * bin_stride ];

    int __shared__ bin_Astart[1];
    int __shared__ bin_Anum[1];
    int __shared__ bin_Bstart[1];
    int __shared__ bin_Bnum[1];
    bool __shared__ samebin[1];
    bool __shared__ end[1];

    int thid = threadIdx.y * blockDim.x + threadIdx.x;
    if (thid == 0)
    {
        *end = false;

        int binA = neighbors[ blockIdx.y * neighb_size1 + 0 ];
        if (binA >= 0){
            *bin_Astart = bin_offsets[ binA ];
            *bin_Anum =   bin_counts[ binA ];
        } else { *end = true; }

        int binB = neighbors[ blockIdx.y * neighb_size1 + blockIdx.x ];
        if (binB >= 0){
            *bin_Bstart = bin_offsets[ binB ];
            *bin_Bnum =   bin_counts[ binB ];
        } else { *end = true; }

        *samebin = binA == binB;
    }
    __syncthreads();
    if ( *end ) { return; }

    LoadBin<scalar_t>(pts, bin_ptids, bin_a, *bin_Astart, *bin_Anum, pt_size1, bin_stride);

    LoadBin<scalar_t>(pts, bin_ptids, bin_b, *bin_Bstart, *bin_Bnum, pt_size1, bin_stride);
    __syncthreads();

    CompareBin(
            edges, glob_counts,
            bin_a, bin_b, *bin_Anum, *bin_Bnum,
            radius, scale_radius,

            bin_stride, *samebin
    );
}

__host__ void frnn_kern_launch(

    int*    neighbors,
    int*    bin_counts,
    int*    bin_offsets,
    int*    bin_ptids,
    torch::Tensor pts,

    int*    glob_counts,
    long*   edges,

    float radius,
    float scale_radius,

    int max_binsize,
    int bin_stride,

    int neighb_size1,
    int pt_size1,

    dim3 blocks,
    dim3 threads,
    size_t shared
){
    AT_DISPATCH_FLOATING_TYPES_AND(torch::ScalarType::Half, pts.scalar_type(), "frnn_main_kernel", ([&] {
        frnn_main_kernel<<<blocks, threads, shared>>>(

            neighbors,
            bin_counts,
            bin_offsets,
            bin_ptids,
            pts.data_ptr<scalar_t>(),

            glob_counts,
            edges,

            radius,
            scale_radius,

            max_binsize,
            bin_stride,

            neighb_size1,
            pt_size1
        );
    }));

}
