#include "hip/hip_runtime.h"
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

#include <vector>

// #include <iostream>
#include <string>
#include <math.h>
#include <stdio.h>

using namespace std;

namespace {

const int n_threadsx = 16;
const int n_threadsy = 16;
const int n_threads2D = n_threadsx * n_threadsy;

const int n_max_pts_bin = 1200;
const int pt_size = 4;

const int edge_size = 2;
const int max_n_edges_buff_per_thread = 4;
const int size_e_buff_per_thread = max_n_edges_buff_per_thread * edge_size;
const int max_n_edges_buff = n_threads2D * max_n_edges_buff_per_thread;
const int size_e_buff = max_n_edges_buff * edge_size;

const int offset_x = 0;
const int offset_y = 1;
const int offset_z = 2;
const int offset_s = 3;
// const int wait_count = 1e7;



__global__ void frnn_cuda_forward_kernel(
    const int* neighbor_bins,
    const float* pts,
    const int* pt_idxs,
    const int* first_pt_idxs,
    const float radius,
    const float scale_radius,
    const int n_max_neighbors,
    int* edges,
    int* i_edges,
    const int max_size_edges
   ) {

    int i_bin_a = blockIdx.x;
    int idx_i_bin_b = blockIdx.y;
    int i_bin_b = neighbor_bins[i_bin_a * n_max_neighbors + idx_i_bin_b];

    // if the bin is empty:
    if (first_pt_idxs[i_bin_a] == -1) {
        return;
    }

    if (first_pt_idxs[i_bin_b] == -1) {
        return;
    }

    // neighboring bins in the matrix that are empty
    // should have been set to -1
    // but there might be more bin_b's that 
    // are not -1 after a bin_b that is -1 . . .
    if (i_bin_b == -1) {return;}

    // don't double check bin pairs
    if (i_bin_b < i_bin_a) {return;}



    // assert(blockDim.x == 1);

    // stores the points for bin_a
    __shared__ float bin_a[n_max_pts_bin * pt_size];
    __shared__ float bin_b[n_max_pts_bin * pt_size];
    // stores the pt ids for bin_a
    __shared__ int bin_a_ids[n_max_pts_bin];
    __shared__ int bin_b_ids[n_max_pts_bin];
    __shared__ int edge_buff[size_e_buff];


    const int i_thread2D = threadIdx.x * blockDim.y + threadIdx.y;

    const int i_buff_0 = i_thread2D * size_e_buff_per_thread;
    int i_buff = i_buff_0;
    int full_i_buff = i_buff_0 + size_e_buff_per_thread;

    //////////////
    // load bin_a
    //////////////
    if (threadIdx.x == 0 && threadIdx.y == 0) {

        bool set_neg_1 = false;
        int inext = first_pt_idxs[i_bin_a];
        for (int i=0; i < n_max_pts_bin; i++) {

            if (set_neg_1) {
                bin_a_ids[i] = -1;
                continue;
            }

            bin_a_ids[i] = inext;

            if (inext == -1) {
                set_neg_1 = true;
                continue;
            }

            int i_pts_start = inext * pt_size;
            int i_bin = i * pt_size;
            bin_a[i_bin + offset_x] = pts[i_pts_start + offset_x];
            bin_a[i_bin + offset_y] = pts[i_pts_start + offset_y];
            bin_a[i_bin + offset_z] = pts[i_pts_start + offset_z];
            bin_a[i_bin + offset_s] = pts[i_pts_start + offset_s];

            inext = pt_idxs[inext];
        }

        /*----------
        LOAD BIN B
        -----------*/
        set_neg_1 = false;
        inext = first_pt_idxs[i_bin_b];
        for (int i=0; i < n_max_pts_bin; i++) {
            
            if (set_neg_1) {
                bin_b_ids[i] = -1;
                continue;
            }

            bin_b_ids[i] = inext;

            if (inext == -1) {
                set_neg_1 = true;
                continue;
            }

            int i_pts_start = inext * pt_size;
            int i_bin = i * pt_size;
            bin_b[i_bin + offset_x] = pts[i_pts_start + offset_x];
            bin_b[i_bin + offset_y] = pts[i_pts_start + offset_y];
            bin_b[i_bin + offset_z] = pts[i_pts_start + offset_z];
            bin_b[i_bin + offset_s] = pts[i_pts_start + offset_s];

            inext = pt_idxs[inext];
        }

    }

    __syncthreads();

    /*---------------------
    THE COMPARISIONS
    now do the comparison between 
    bin_a's pts and bin_b's pts
    ----------------------*/

    // ia is the bin index for the current pt a
    // so it is NOT the index into the pts matrix
    for (int ia = threadIdx.x; ia < n_max_pts_bin; ia+=blockDim.x) {

        if (ia >= n_max_pts_bin) {break;}

        if (bin_a_ids[ia] <= -1) {break;}

        float ax = bin_a[ia * pt_size + offset_x];
        float ay = bin_a[ia * pt_size + offset_y];
        float az = bin_a[ia * pt_size + offset_z];
        float as = bin_a[ia * pt_size + offset_s];

        for (int ib = threadIdx.y; ib < n_max_pts_bin; ib+=blockDim.y) {

            if (ib >= n_max_pts_bin) {break;}

            if (bin_b_ids[ib] <= -1) {break;}

            // don't compare the same point to itself:
            if (bin_b_ids[ib] == bin_a_ids[ia]) {continue;}

            // if it's the same bin,
            // only compare lower points to higher points
            if ((i_bin_a == i_bin_b) && (bin_b_ids[ib] <= bin_a_ids[ia])) {
                continue;
            }

            float bx = bin_b[ib * pt_size + offset_x];
            float by = bin_b[ib * pt_size + offset_y];
            float bz = bin_b[ib * pt_size + offset_z];
            float bs = bin_b[ib * pt_size + offset_s];

            // check that the scales are close enough
            float scale_max = as;
            float scale_min = bs;
            if (as < bs) {
                scale_max = bs;
                scale_min = as;
            }

            if ((scale_max / scale_min) > scale_radius) {continue;}

            float diffx = bx - ax;
            float diffy = by - ay;
            float diffz = bz - az;
            float dist = diffx * diffx + diffy * diffy + diffz * diffz;
            dist = sqrt(dist);

            float log_avg_scale = sqrt(as * bs);
            if (dist > radius / log_avg_scale) {continue;}

            // printf("blx %d bly %d thx %d thy %d ia %d ib %d \n",
            //     blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, bin_a_ids[ia], bin_b_ids[ib]);

            // now we can add the edge to the edge buffer
            edge_buff[i_buff + 0] = bin_a_ids[ia];
            edge_buff[i_buff + 1] = bin_b_ids[ib];

            i_buff += edge_size;

            /*---------------------------
            empty the buffer in the loop:
            -----------------------------*/
            if (i_buff == full_i_buff) {
                
                // empty the buffer and put it in the edges array
                int this_i_edges = atomicAdd(&i_edges[0], size_e_buff_per_thread);

                // make sure we don't overfill the edge output tensor
                int e_stop = size_e_buff_per_thread;
                if (this_i_edges + size_e_buff_per_thread > max_size_edges) {
                    e_stop = max_size_edges - this_i_edges;
                }

                for (int ie = 0; ie < e_stop; ie ++) {
                    edges[this_i_edges + ie] = edge_buff[i_buff_0 + ie];
                }

                i_buff = i_buff_0;
            }
        }
    }

    // __syncthreads();
    /*---------------------------------------------
    empty the buffer one more time at the very end:
    -----------------------------------------------*/

    // if (threadIdx.x == 0 && threadIdx.y == 0) {

    if (i_buff > i_buff_0) {

        // printf("blx %d bly %d thx %d thy %d i_buff %d i_buff_0 %d\n",
        //     blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, i_buff, i_buff_0);

        int this_size_e_buff_per_thread = i_buff - i_buff_0;

        int this_i_edges = atomicAdd(&i_edges[0], this_size_e_buff_per_thread);

        // make sure we don't overfill the edge output tensor
        int e_stop = this_size_e_buff_per_thread;
        if (this_i_edges + this_size_e_buff_per_thread > max_size_edges) {
            e_stop = max_size_edges - this_i_edges;
        }

        for (int ie = 0; ie < e_stop; ie ++) {
            // printf("blx %d bly %d thx %d thy %d \n  i_buff %d i_buff_0 %d ie %d id%d\n",
            // blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, 
            // i_buff, i_buff_0, ie, edge_buff[ie]);
            edges[this_i_edges + ie] = edge_buff[i_buff_0 + ie];
        }
    }
    // }

    return;
}

} // closing tag for <namespace> 




std::vector<at::Tensor> frnn_cuda_forward(
        at::Tensor neighbor_bins,
        at::Tensor pts,
        at::Tensor pt_idxs,
        at::Tensor first_pt_idxs,
        float radius,
        float scale_radius,
        at::Tensor edges,
        at::Tensor i_edges
       ) {

    const dim3 threads(n_threadsx, n_threadsy);

    const int n_bins = neighbor_bins.size(0);
    const int n_max_neighbors = neighbor_bins.size(1);
    const dim3 blocks(n_bins, n_max_neighbors);
    
    const int max_size_edges = edges.size(0) * edges.size(1);
  
    AT_DISPATCH_INTEGRAL_TYPES(edges.type(), "frnn_forward_cuda", ([&] {
    frnn_cuda_forward_kernel<<<blocks, threads>>>(
        neighbor_bins.data<int>(),
        pts.data<float>(),
        pt_idxs.data<int>(),
        first_pt_idxs.data<int>(),
        radius,
        scale_radius,
        n_max_neighbors,
        edges.data<int>(),
        i_edges.data<int>(),
        max_size_edges);
    }));

    // return {edges.slice(0, i_edges.data<int>()[0]), i_edges};
    return {edges, i_edges};
}
